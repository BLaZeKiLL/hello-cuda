#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "utils.cuh"

__device__ int mandelbrot(float x0, float y0, int max_iter) {
    float x = 0.0f, y = 0.0f;
    int iter = 0;

    while (x * x + y * y <= 4.0f && iter < max_iter) {
        float x_temp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = x_temp;
        iter++;
    }
    return iter;
}

__global__ void mandelbrot_kernel(unsigned char* image, int width, int height, float x_min, float x_max, float y_min, float y_max, int max_iter) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < width && idy < height) {
        // Map pixel coordinates to complex plane
        float x0 = x_min + (x_max - x_min) * idx / width;
        float y0 = y_min + (y_max - y_min) * idy / height;

        // Compute Mandelbrot iterations
        int iter = mandelbrot(x0, y0, max_iter);

        // Normalize iteration count to [0, 1]
        float hue = (float)iter / max_iter;

        // Apply HSV to RGB conversion (use a fixed saturation and value)
        unsigned char r, g, b;
        if (iter < max_iter) {
            hsv_to_rgb(hue, 0.8f, 1.0f, r, g, b);
        } else {
            r = g = b = 0;  // Points that don't escape are black
        }

        // Write color to image
        int pixel_idx = (idy * width + idx) * 3;
        image[pixel_idx + 0] = r;  // Red
        image[pixel_idx + 1] = g;  // Green
        image[pixel_idx + 2] = b;  // Blue
    }
}

void execute(unsigned char* h_image, float x_min, float x_max, float y_min, float y_max, int width, int height, int max_iter) {
    const int image_size = width * height * 3;

    unsigned char* d_image;

    // Allocate memory on the GPU
    hipMalloc(&d_image, image_size);

    // Define grid and block dimensions
    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);

    // Launch the Mandelbrot kernel
    mandelbrot_kernel<<<grid_size, block_size>>>(d_image, width, height, x_min, x_max, y_min, y_max, max_iter);
    hipDeviceSynchronize();

    // Copy result back to the host
    hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_image);
}